#include<stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <ctype.h>
#include <time.h>
#define MAX_LENGTH 10000
#define REPEAT_TIMES 10
#define MAX_KEY 10
#define NUM_LETTERS 26 



__constant__ char d_inputString[MAX_LENGTH]; 
__device__ char d_key[10]; 
__device__ char d_key_calculated[1024]; 


__device__ bool is_upper(char c) {
    return c >= 'A' && c <= 'Z';
}


__global__ void repeatString(char *output, int inputLength){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int offset = idx * inputLength;
    for(int i = 0; i<inputLength;i++)
        output[offset+i]=d_inputString[i];
}

__global__ void vignere_encrypt_cuda(const char* msg, char* encrypted, int msgLen, int keyLen) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < msgLen) {
        if (is_upper(msg[idx])) {
            int msgIndex = msg[idx] - 'A';
            int keyIndex = d_key[idx % keyLen] - 'A';
            int encryptedIndex = (msgIndex + keyIndex) % NUM_LETTERS;
            encrypted[idx] = 'A' + encryptedIndex;
        } else {
            encrypted[idx] = msg[idx];
        }
    }
}

__global__ void crack_vigenere_kernel(const char* cipher, int cipher_len, int n, char* keys) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        int freq[NUM_LETTERS] = {0};
        for (int j = i; j < cipher_len; j += n) {
            if (is_upper(cipher[j])) {
                freq[cipher[j] - 'A']++;
            }
        }
        
        int max_freq = 0;
        char max_char = 0;
        for (int k = 0; k < NUM_LETTERS; k++) {
            if (freq[k] > max_freq) {
                max_freq = freq[k];
                max_char = k + 'A';
            }
        }
        
        keys[i] = ((max_char - 'E' + NUM_LETTERS) % NUM_LETTERS) + 'A';
    }
}
__global__ void vignere_decrypt_cuda(const char* encrypted, char* decrypted, int msgLen, int keyLen) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < msgLen) {
        if (is_upper(encrypted[idx])) {
            int encryptedIndex = encrypted[idx] - 'A';
            int keyIndex = d_key[idx % keyLen] - 'A'; 
            int msgIndex = (encryptedIndex - keyIndex + NUM_LETTERS) % NUM_LETTERS;
            decrypted[idx] = 'A' + msgIndex;
        } else {
            decrypted[idx] = encrypted[idx];
        }
    }
}


char* only_alphabets(const char* text) {
    char* res = (char*)malloc(strlen(text) + 1);
    if (res == NULL) {
        printf("Memory allocation failed.\n");
        exit(1);
    }

    int j = 0;
    for (int i = 0; text[i] != '\0'; i++) {
        if (isalpha((unsigned char)text[i])) 
        {
            res[j++] = toupper((unsigned char)text[i]);
        }
    }
    res[j] = '\0'; 

    return res;
}

char* process_string_CUDA(char* temp) {
    temp = only_alphabets(temp);
    char *d_output;
    int inputLength = strlen(temp);

    size_t outputSize = MAX_LENGTH * REPEAT_TIMES; 
    hipMemcpyToSymbol(HIP_SYMBOL(d_inputString), temp, inputLength + 1);
    hipDeviceSynchronize(); 

    hipMalloc(&d_output, outputSize);
    hipMemset(d_output, 0, outputSize);
    repeatString<<<REPEAT_TIMES, 1>>>(d_output, inputLength); 
    hipDeviceSynchronize();

    char *plainTextInput = (char*) malloc(outputSize);
    hipMemcpy(plainTextInput, d_output, outputSize, hipMemcpyDeviceToHost);
    hipFree(d_output);
    free(temp);
    return plainTextInput; 
}

char *cudaEncrypt(char *processedInput,char *key)
{
    char* d_message;
    char* d_encrypted;
    char* d_decrypted;


    int msgLen=strlen(processedInput);
    int keyLen = strlen(key);
    
    hipMalloc((void**)&d_message, msgLen+1);
    hipMalloc((void**)&d_encrypted, msgLen+1);
    hipMalloc((void**)&d_decrypted, msgLen+1); 
    
    hipMemcpy(d_message, processedInput, msgLen+1, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(d_key), key, keyLen+1); 
    
    int blockSize = 256; 
    int numBlocks = (msgLen + blockSize - 1) / blockSize; 
    
    vignere_encrypt_cuda<<<numBlocks, blockSize>>>(d_message, d_encrypted, msgLen, keyLen);

    char* encrypted = (char*)malloc(msgLen+1);
    hipMemcpy(encrypted, d_encrypted, msgLen+1, hipMemcpyDeviceToHost);
    hipFree(d_message);
    hipFree(d_encrypted);
    hipFree(d_decrypted);
    return encrypted;

}
__global__ void calculate_matches(const char *cipher, int length, int *matches) {
    int shift = blockIdx.x * blockDim.x + threadIdx.x;
    if (shift >= length) return;

    int matchCount = 0;
    for (int i = 0; i < length; ++i) {
        if (cipher[i] == cipher[(i - shift + length) % length]) {
            ++matchCount;
        }
    }
    matches[shift] = matchCount;
}

double mean(const int arr[], int size) {
    double sum = 0;
    for (int i = 0; i < size; i++) {
        sum += arr[i];
    }
    return sum / size;
}
double variance(const int arr[], int size, double mean) {
    double sum = 0;
    for (int i = 0; i < size; i++) {
        sum += (arr[i] - mean) * (arr[i] - mean);
    }
    return sum / size;
}
int mode(const int arr[], int size) {
    int maxValue = 0, maxCount = 0;

    for (int i = 0; i < size; ++i) {
        int count = 0;
        for (int j = 0; j < size; ++j) {
            if (arr[j] == arr[i]) ++count;
        }
        
        if (count > maxCount) {
            maxCount = count;
            maxValue = arr[i];
        }
    }
    return maxValue;
}

int guess_key_length_cuda(const char *cipher) {
    int length = strlen(cipher);
    char *d_cipher;
    int *d_matches, *matches;

    hipMalloc((void **)&d_cipher, length);
    hipMalloc((void **)&d_matches, length * sizeof(int));

    hipMemcpy(d_cipher, cipher, length, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocks = (length + threadsPerBlock - 1) / threadsPerBlock;

    calculate_matches<<<blocks, threadsPerBlock>>>(d_cipher, length, d_matches);

    matches = (int *)malloc(length * sizeof(int));
    hipMemcpy(matches, d_matches, length * sizeof(int), hipMemcpyDeviceToHost);




     double dat_means = mean(matches + 1, length - 1);
    double dat_std_dev = sqrt(variance(matches + 1, length - 1, dat_means));

    int* peaks = (int*)malloc(length * sizeof(int));
    int peaks_count = 0;
    for (int i = 0; i < length; i++) {
        if (matches[i] >= dat_std_dev + dat_means) {
            peaks[peaks_count++] = i;
        }
    }

    int* peak_diff = (int*)malloc((peaks_count - 1) * sizeof(int));
    for (int i = 0; i < peaks_count - 1; i++) {
        peak_diff[i] = peaks[i + 1] - peaks[i];
    }

    int key_length = mode(peak_diff, peaks_count - 1);



    hipFree(d_cipher);
    hipFree(d_matches);
    free(matches); 
    free(peaks);
    free(peak_diff);

    return key_length;
}

char* crack_vigenere_cuda(const char* cipher, int n) {
    int cipher_len = strlen(cipher);
    char* dev_cipher = NULL;
    char* dev_keys = NULL;
    char* keys = (char*)malloc(n + 1);
    
    hipMalloc((void**)&dev_cipher, cipher_len * sizeof(char));
    hipMalloc((void**)&dev_keys, n * sizeof(char));
    
    hipMemcpy(dev_cipher, cipher, cipher_len * sizeof(char), hipMemcpyHostToDevice);
    
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    crack_vigenere_kernel<<<blocksPerGrid, threadsPerBlock>>>(dev_cipher, cipher_len, n, dev_keys);
    
    hipMemcpy(keys, dev_keys, n * sizeof(char), hipMemcpyDeviceToHost);
    keys[n] = '\0';
    
    hipFree(dev_cipher);
    hipFree(dev_keys);
    
    return keys;
}
char* cudaDecrypt(char *encrypted, char *key) {
    char* d_encrypted;
    char* d_decrypted;
    int msgLen = strlen(encrypted);
    int keyLen = strlen(key);

    hipMalloc((void**)&d_encrypted, msgLen + 1);
    hipMalloc((void**)&d_decrypted, msgLen + 1);

    hipMemcpy(d_encrypted, encrypted, msgLen + 1, hipMemcpyHostToDevice);

    hipMemcpyToSymbol(HIP_SYMBOL(d_key_calculated), key, keyLen + 1);

    int blockSize = 256;
    int numBlocks = (msgLen + blockSize - 1) / blockSize;
    vignere_decrypt_cuda<<<numBlocks, blockSize>>>(d_encrypted, d_decrypted, msgLen, keyLen);

    hipDeviceSynchronize();

    char* decrypted = (char*)malloc(msgLen + 1);

    hipMemcpy(decrypted, d_decrypted, msgLen + 1, hipMemcpyDeviceToHost);

    hipFree(d_encrypted);
    hipFree(d_decrypted);
    return decrypted;
}




int main ()
{
    clock_t start = clock();


    char inputString[MAX_LENGTH]="Standard deviation is calculated as the square root of the variance Alternatively it is calculated by finding the mean of a data set finding the difference of each data point to the mean, squaring the differences adding them together dividing by the number of points in the data set less and finding the square root. Standard deviation is important because it can help users assess risk Consider an investment option with an average annual return of 10 per year However this average was derived from the past three year returns of many values By calculating the standard deviation and understanding your low likelihood of actually averaging 10 in any single given year your better armed to make informed decisions and recognizing underlying risk";
    char key[MAX_KEY]="SUJ";
    char * processedInput = process_string_CUDA(inputString);
    char * encrypted = cudaEncrypt(processedInput,key);
    int n = guess_key_length_cuda(encrypted);
    char* key_calculated = crack_vigenere_cuda(encrypted,n);

    char *calculatedPlainText=cudaDecrypt(encrypted,key_calculated);
    printf("%s",key_calculated);
    clock_t end = clock();

    // Calculate the time spent
    double time_spent = (double)(end - start) / CLOCKS_PER_SEC;
    printf("Time spent: %f seconds\n", time_spent);


return 0;
}
